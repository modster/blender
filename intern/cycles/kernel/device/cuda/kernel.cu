
#include <hip/hip_runtime.h>
/*
 * Copyright 2011-2013 Blender Foundation
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

/* CUDA kernel entry points */

#ifdef __CUDA_ARCH__

#  include "kernel/device/cuda/compat.h"
#  include "kernel/device/cuda/config.h"
#  include "kernel/device/cuda/globals.h"
#  include "kernel/device/cuda/image.h"
#  include "kernel/device/cuda/parallel_active_index.h"
#  include "kernel/device/cuda/parallel_prefix_sum.h"
#  include "kernel/device/cuda/parallel_sorted_index.h"

#  include "kernel/integrator/integrator_path_state.h"
#  include "kernel/integrator/integrator_state.h"
#  include "kernel/integrator/integrator_state_util.h"

#  include "kernel/integrator/integrator_init_from_camera.h"
#  include "kernel/integrator/integrator_intersect_closest.h"
#  include "kernel/integrator/integrator_intersect_shadow.h"
#  include "kernel/integrator/integrator_intersect_subsurface.h"
#  include "kernel/integrator/integrator_megakernel.h"
#  include "kernel/integrator/integrator_shade_background.h"
#  include "kernel/integrator/integrator_shade_light.h"
#  include "kernel/integrator/integrator_shade_shadow.h"
#  include "kernel/integrator/integrator_shade_surface.h"
#  include "kernel/integrator/integrator_shade_volume.h"

#  include "kernel/kernel_adaptive_sampling.h"
#  include "kernel/kernel_bake.h"
#  include "kernel/kernel_film.h"
#  include "kernel/kernel_work_stealing.h"

/* TODO: move cryptomatte post sorting to its own kernel. */
#  if 0
/* kernels */
extern "C" __global__ void CUDA_LAUNCH_BOUNDS(CUDA_KERNEL_BLOCK_NUM_THREADS, CUDA_KERNEL_MAX_REGISTERS)
    kernel_cuda_path_trace(KernelWorkTile *tile, uint work_size)
{
	int work_index = ccl_global_id(0);
	bool thread_is_active = work_index < work_size;
	uint x, y, sample;
	KernelGlobals kg;
	if(thread_is_active) {
		get_work_pixel(tile, work_index, &x, &y, &sample);

		kernel_path_trace(&kg, tile->buffer, sample, x, y, tile->offset, tile->stride);
	}

	if(kernel_data.film.cryptomatte_passes) {
		__syncthreads();
		if(thread_is_active) {
			kernel_cryptomatte_post(&kg, tile->buffer, sample, x, y, tile->offset, tile->stride);
		}
	}
}
#  endif

/* --------------------------------------------------------------------
 * Integrator.
 */

extern "C" __global__ void CUDA_LAUNCH_BOUNDS(CUDA_KERNEL_BLOCK_NUM_THREADS,
                                              CUDA_KERNEL_MAX_REGISTERS)
    kernel_cuda_integrator_init_from_camera(const int *path_index_array,
                                            KernelWorkTile *tile,
                                            float *render_buffer,
                                            const int tile_work_size,
                                            const int path_index_offset)
{
  const int global_index = ccl_global_id(0);
  const int work_index = global_index;
  bool thread_is_active = work_index < tile_work_size;
  if (thread_is_active) {
    const int path_index = (path_index_array) ? path_index_array[global_index] :
                                                path_index_offset + global_index;

    uint x, y, sample;
    get_work_pixel(tile, work_index, &x, &y, &sample);
    integrator_init_from_camera(NULL, path_index, tile, render_buffer, x, y, sample);
  }
}

extern "C" __global__ void CUDA_LAUNCH_BOUNDS(CUDA_KERNEL_BLOCK_NUM_THREADS,
                                              CUDA_KERNEL_MAX_REGISTERS)
    kernel_cuda_integrator_intersect_closest(const int *path_index_array, const int work_size)
{
  const int global_index = ccl_global_id(0);

  if (global_index < work_size) {
    const int path_index = (path_index_array) ? path_index_array[global_index] : global_index;
    integrator_intersect_closest(NULL, path_index);
  }
}

extern "C" __global__ void CUDA_LAUNCH_BOUNDS(CUDA_KERNEL_BLOCK_NUM_THREADS,
                                              CUDA_KERNEL_MAX_REGISTERS)
    kernel_cuda_integrator_intersect_shadow(const int *path_index_array, const int work_size)
{
  const int global_index = ccl_global_id(0);

  if (global_index < work_size) {
    const int path_index = (path_index_array) ? path_index_array[global_index] : global_index;
    integrator_intersect_shadow(NULL, path_index);
  }
}

extern "C" __global__ void CUDA_LAUNCH_BOUNDS(CUDA_KERNEL_BLOCK_NUM_THREADS,
                                              CUDA_KERNEL_MAX_REGISTERS)
    kernel_cuda_integrator_intersect_subsurface(const int *path_index_array, const int work_size)
{
  const int global_index = ccl_global_id(0);

  if (global_index < work_size) {
    const int path_index = (path_index_array) ? path_index_array[global_index] : global_index;
    integrator_intersect_subsurface(NULL, path_index);
  }
}

extern "C" __global__ void CUDA_LAUNCH_BOUNDS(CUDA_KERNEL_BLOCK_NUM_THREADS,
                                              CUDA_KERNEL_MAX_REGISTERS)
    kernel_cuda_integrator_shade_background(const int *path_index_array,
                                            float *render_buffer,
                                            const int work_size)
{
  const int global_index = ccl_global_id(0);

  if (global_index < work_size) {
    const int path_index = (path_index_array) ? path_index_array[global_index] : global_index;
    integrator_shade_background(NULL, path_index, render_buffer);
  }
}

extern "C" __global__ void CUDA_LAUNCH_BOUNDS(CUDA_KERNEL_BLOCK_NUM_THREADS,
                                              CUDA_KERNEL_MAX_REGISTERS)
    kernel_cuda_integrator_shade_light(const int *path_index_array,
                                       float *render_buffer,
                                       const int work_size)
{
  const int global_index = ccl_global_id(0);

  if (global_index < work_size) {
    const int path_index = (path_index_array) ? path_index_array[global_index] : global_index;
    integrator_shade_light(NULL, path_index, render_buffer);
  }
}

extern "C" __global__ void CUDA_LAUNCH_BOUNDS(CUDA_KERNEL_BLOCK_NUM_THREADS,
                                              CUDA_KERNEL_MAX_REGISTERS)
    kernel_cuda_integrator_shade_shadow(const int *path_index_array,
                                        float *render_buffer,
                                        const int work_size)
{
  const int global_index = ccl_global_id(0);

  if (global_index < work_size) {
    const int path_index = (path_index_array) ? path_index_array[global_index] : global_index;
    integrator_shade_shadow(NULL, path_index, render_buffer);
  }
}

extern "C" __global__ void CUDA_LAUNCH_BOUNDS(CUDA_KERNEL_BLOCK_NUM_THREADS,
                                              CUDA_KERNEL_MAX_REGISTERS)
    kernel_cuda_integrator_shade_surface(const int *path_index_array,
                                         float *render_buffer,
                                         const int work_size)
{
  const int global_index = ccl_global_id(0);

  if (global_index < work_size) {
    const int path_index = (path_index_array) ? path_index_array[global_index] : global_index;
    integrator_shade_surface(NULL, path_index, render_buffer);
  }
}

extern "C" __global__ void CUDA_LAUNCH_BOUNDS(CUDA_KERNEL_BLOCK_NUM_THREADS,
                                              CUDA_KERNEL_MAX_REGISTERS)
    kernel_cuda_integrator_shade_volume(const int *path_index_array,
                                        float *render_buffer,
                                        const int work_size)
{
  const int global_index = ccl_global_id(0);

  if (global_index < work_size) {
    const int path_index = (path_index_array) ? path_index_array[global_index] : global_index;
    integrator_shade_volume(NULL, path_index, render_buffer);
  }
}

extern "C" __global__ void CUDA_LAUNCH_BOUNDS(CUDA_KERNEL_BLOCK_NUM_THREADS,
                                              CUDA_KERNEL_MAX_REGISTERS)
    kernel_cuda_integrator_megakernel(const int *path_index_array,
                                      float *render_buffer,
                                      const int work_size)
{
  const int global_index = ccl_global_id(0);

  if (global_index < work_size) {
    const int path_index = (path_index_array) ? path_index_array[global_index] : global_index;
    integrator_megakernel(NULL, path_index, render_buffer);
  }
}

extern "C" __global__ void __launch_bounds__(CUDA_PARALLEL_ACTIVE_INDEX_DEFAULT_BLOCK_SIZE)
    kernel_cuda_integrator_queued_paths_array(int num_states,
                                              int *indices,
                                              int *num_indices,
                                              int kernel)
{
  cuda_parallel_active_index_array<CUDA_PARALLEL_ACTIVE_INDEX_DEFAULT_BLOCK_SIZE>(
      num_states, indices, num_indices, [kernel](const int path_index) {
        return (INTEGRATOR_STATE(path, queued_kernel) == kernel);
      });
}

extern "C" __global__ void __launch_bounds__(CUDA_PARALLEL_ACTIVE_INDEX_DEFAULT_BLOCK_SIZE)
    kernel_cuda_integrator_queued_shadow_paths_array(int num_states,
                                                     int *indices,
                                                     int *num_indices,
                                                     int kernel)
{
  cuda_parallel_active_index_array<CUDA_PARALLEL_ACTIVE_INDEX_DEFAULT_BLOCK_SIZE>(
      num_states, indices, num_indices, [kernel](const int path_index) {
        return (INTEGRATOR_STATE(shadow_path, queued_kernel) == kernel);
      });
}

extern "C" __global__ void __launch_bounds__(CUDA_PARALLEL_ACTIVE_INDEX_DEFAULT_BLOCK_SIZE)
    kernel_cuda_integrator_terminated_paths_array(int num_states,
                                                  int *indices,
                                                  int *num_indices,
                                                  int unused_kernel)
{
  cuda_parallel_active_index_array<CUDA_PARALLEL_ACTIVE_INDEX_DEFAULT_BLOCK_SIZE>(
      num_states, indices, num_indices, [](const int path_index) {
        return (INTEGRATOR_STATE(path, queued_kernel) == 0) &&
               (INTEGRATOR_STATE(shadow_path, queued_kernel) == 0);
      });
}

extern "C" __global__ void __launch_bounds__(CUDA_PARALLEL_SORTED_INDEX_DEFAULT_BLOCK_SIZE)
    kernel_cuda_integrator_sorted_paths_array(
        int num_states, int *indices, int *num_indices, int *key_prefix_sum, int kernel)
{
  cuda_parallel_sorted_index_array<CUDA_PARALLEL_SORTED_INDEX_DEFAULT_BLOCK_SIZE>(
      num_states, indices, num_indices, key_prefix_sum, [kernel](const int path_index) {
        return (INTEGRATOR_STATE(path, queued_kernel) == kernel) ?
                   __integrator_sort_key[path_index] :
                   CUDA_PARALLEL_SORTED_INDEX_INACTIVE_KEY;
      });
}

extern "C" __global__ void __launch_bounds__(CUDA_PARALLEL_PREFIX_SUM_DEFAULT_BLOCK_SIZE)
    kernel_cuda_prefix_sum(int *values, int num_values)
{
  cuda_parallel_prefix_sum<CUDA_PARALLEL_PREFIX_SUM_DEFAULT_BLOCK_SIZE>(values, num_values);
}

/* --------------------------------------------------------------------
 * Adaptive sampling.
 */

extern "C" __global__ void CUDA_LAUNCH_BOUNDS(CUDA_KERNEL_BLOCK_NUM_THREADS,
                                              CUDA_KERNEL_MAX_REGISTERS)
    kernel_cuda_adaptive_sampling_convergence_check(
        float *render_buffer, int sx, int sy, int sw, int sh, int sample, int offset, int stride)
{
  const int work_index = ccl_global_id(0);
  const int y = work_index / sw;
  const int x = work_index - y * sw;

  if (x < sw && y < sh) {
    kernel_adaptive_sampling_convergence_check(
        NULL, render_buffer, sx + x, sy + y, sample, offset, stride);
  }
}

extern "C" __global__ void CUDA_LAUNCH_BOUNDS(CUDA_KERNEL_BLOCK_NUM_THREADS,
                                              CUDA_KERNEL_MAX_REGISTERS)
    kernel_cuda_adaptive_sampling_filter_x(
        float *render_buffer, int sx, int sy, int sw, int sh, int offset, int stride)
{
  const int y = ccl_global_id(0);

  if (y < sh) {
    kernel_adaptive_sampling_filter_x(NULL, render_buffer, sy + y, sx, sw, offset, stride);
  }
}

extern "C" __global__ void CUDA_LAUNCH_BOUNDS(CUDA_KERNEL_BLOCK_NUM_THREADS,
                                              CUDA_KERNEL_MAX_REGISTERS)
    kernel_cuda_adaptive_sampling_filter_y(
        float *render_buffer, int sx, int sy, int sw, int sh, int offset, int stride)
{
  const int x = ccl_global_id(0);

  if (x < sw) {
    kernel_adaptive_sampling_filter_y(NULL, render_buffer, sx + x, sy, sh, offset, stride);
  }
}

/* --------------------------------------------------------------------
 * Film.
 */

/* Convert to Display Buffer */

extern "C" __global__ void CUDA_LAUNCH_BOUNDS(CUDA_KERNEL_BLOCK_NUM_THREADS,
                                              CUDA_KERNEL_MAX_REGISTERS)
    kernel_cuda_convert_to_half_float(uchar4 *rgba,
                                      float *render_buffer,
                                      float sample_scale,
                                      int sx,
                                      int sy,
                                      int sw,
                                      int sh,
                                      int offset,
                                      int stride)
{
  const int work_index = ccl_global_id(0);
  const int y = work_index / sw;
  const int x = work_index - y * sw;

  if (x < sw && y < sh) {
    kernel_film_convert_to_half_float(
        NULL, rgba, render_buffer, sample_scale, sx + x, sy + y, offset, stride);
  }
}

/* --------------------------------------------------------------------
 * Shader evaluaiton.
 */

/* Displacement */

extern "C" __global__ void CUDA_LAUNCH_BOUNDS(CUDA_KERNEL_BLOCK_NUM_THREADS,
                                              CUDA_KERNEL_MAX_REGISTERS)
    kernel_cuda_shader_eval_displace(KernelShaderEvalInput *input,
                                     float4 *output,
                                     const int offset,
                                     const int work_size)
{
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  if (i < work_size) {
    kernel_displace_evaluate(NULL, input, output, offset + i);
  }
}

/* Background Shader Evaluation */

extern "C" __global__ void CUDA_LAUNCH_BOUNDS(CUDA_KERNEL_BLOCK_NUM_THREADS,
                                              CUDA_KERNEL_MAX_REGISTERS)
    kernel_cuda_shader_eval_background(KernelShaderEvalInput *input,
                                       float4 *output,
                                       const int offset,
                                       const int work_size)
{
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  if (i < work_size) {
    kernel_background_evaluate(NULL, input, output, offset + i);
  }
}

/* --------------------------------------------------------------------
 * Baking.
 */

#  ifdef __BAKING__
extern "C" __global__ void CUDA_LAUNCH_BOUNDS(CUDA_KERNEL_BLOCK_NUM_THREADS,
                                              CUDA_KERNEL_MAX_REGISTERS)
    kernel_cuda_bake(KernelWorkTile *tile, uint work_size)
{
  /* TODO */
#    if 0
  int work_index = ccl_global_id(0);

  if (work_index < work_size) {
    uint x, y, sample;
    get_work_pixel(tile, work_index, &x, &y, &sample);

    KernelGlobals kg;
    kernel_bake_evaluate(&kg, tile->buffer, sample, x, y, tile->offset, tile->stride);
  }
#    endif
}
#  endif

/* --------------------------------------------------------------------
 * Denoising.
 */

extern "C" __global__ void CUDA_LAUNCH_BOUNDS(CUDA_KERNEL_BLOCK_NUM_THREADS,
                                              CUDA_KERNEL_MAX_REGISTERS)
    kernel_cuda_filter_convert_to_rgb(float *rgb,
                                      const float *render_buffer,
                                      int sx,
                                      int sy,
                                      int sw,
                                      int sh,
                                      int offset,
                                      int stride,
                                      int pass_stride,
                                      int3 pass_offset,
                                      int num_inputs,
                                      int num_samples)
{
  const int work_index = ccl_global_id(0);
  const int y = work_index / sw;
  const int x = work_index - y * sw;

  if (x < sw && y < sh) {
    const float num_samples_inv = 1.0f / num_samples;

    const int render_pixel_index = offset + (x + sx) + (y + sy) * stride;
    const float *buffer = render_buffer + (uint64_t)render_pixel_index * pass_stride;

    if (num_inputs > 0) {
      const float *in = buffer + pass_offset.x;
      float *out = rgb + (x + y * sw) * 3;
      out[0] = clamp(in[0] * num_samples_inv, 0.0f, 10000.0f);
      out[1] = clamp(in[1] * num_samples_inv, 0.0f, 10000.0f);
      out[2] = clamp(in[2] * num_samples_inv, 0.0f, 10000.0f);
    }

#  if 0
    if (num_inputs > 1) {
      const float *in = buffer + pass_offset.y;
      float *out = rgb + (x + y * sw) * 3 + (sw * sh) * 3;
      out[0] = in[0] * num_samples_inv;
      out[1] = in[1] * num_samples_inv;
      out[2] = in[2] * num_samples_inv;
    }

    if (num_inputs > 2) {
      const float *in = buffer + pass_offset.y;
      float *out = rgb + (x + y * sw) * 3 + (sw * sh * 2) * 3;
      out[0] = in[0] * num_samples_inv;
      out[1] = in[1] * num_samples_inv;
      out[2] = in[2] * num_samples_inv;
    }
#  endif
  }
}

extern "C" __global__ void CUDA_LAUNCH_BOUNDS(CUDA_KERNEL_BLOCK_NUM_THREADS,
                                              CUDA_KERNEL_MAX_REGISTERS)
    kernel_cuda_filter_convert_from_rgb(const float *rgb,
                                        float *render_buffer,
                                        int sx,
                                        int sy,
                                        int sw,
                                        int sh,
                                        int offset,
                                        int stride,
                                        int pass_stride,
                                        int num_samples)
{
  const int work_index = ccl_global_id(0);
  const int y = work_index / sw;
  const int x = work_index - y * sw;

  if (x < sw && y < sh) {
    const float *in = rgb + (x + y * sw) * 3;

    const int render_pixel_index = offset + (x + sx) + (y + sy) * stride;
    float *buffer = render_buffer + (uint64_t)render_pixel_index * pass_stride;

    buffer[0] = in[0] * num_samples;
    buffer[1] = in[1] * num_samples;
    buffer[2] = in[2] * num_samples;
  }
}

#endif
